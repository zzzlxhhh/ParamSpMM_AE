#include "sddmm.h"
#define CUSPARSE_CHECK(func)                                                         \
    {                                                                                \
        hipsparseStatus_t status = (func);                                            \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                       \
        {                                                                            \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, \
                   hipsparseGetErrorString(status), status);                          \
            exit(EXIT_FAILURE);                                                      \
        }                                                                            \
    }
#define CUDA_CHECK(func)                                                                           \
    {                                                                                              \
        hipError_t status = (func);                                                               \
        if (status != hipSuccess)                                                                 \
        {                                                                                          \
            printf("CUDA API failed at file %s line %d with error: %s (%d)\n", __FILE__, __LINE__, \
                   hipGetErrorString(status), status);                                            \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    }

// vendor-provided cusparse sddmm by NVIDIA
torch::Tensor cusparse_sddmm_compute(
    torch::Tensor A, // A_ M x k
    torch::Tensor B, // B_ transpose N x k
    torch::Tensor rowPtr,
    torch::Tensor colIdx,
    torch::Tensor &values)
{
    const int M = A.size(0); // row of A
    const int K = A.size(1); // col of A
    const int N = B.size(0); // col of B
    const int nnz = values.size(0); 

    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t csrDescr;
    hipsparseDnMatDescr_t AMatDecsr, BMatDecsr;
    float alpha = 1.0f, beta = 0.0f;
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    // S
    auto csr_indptr_d = rowPtr.data_ptr<int>();
    auto csr_indices_d = colIdx.data_ptr<int>();
    auto csr_values_d = values.data_ptr<float>();
    // A & B
    auto A_d = A.data_ptr<float>();
    auto B_d = B.data_ptr<float>();

    // // creating sparse csr matrix
    CUSPARSE_CHECK(hipsparseCreateCsr(
        &csrDescr, M, N, nnz, csr_indptr_d, csr_indices_d, csr_values_d,
        HIPSPARSE_INDEX_32I, // index 32-integer for indptr
        HIPSPARSE_INDEX_32I, // index 32-integer for indices
        HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_32F // datatype: 32-bit float real number
        ));

    // creating dense matrices
    CUSPARSE_CHECK(hipsparseCreateDnMat(&AMatDecsr, M, K, K, A_d, HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&BMatDecsr, N, K, K, B_d, HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));

    // allocate workspace buffer
    size_t bufferSize = 0;
    void *dBuffer = NULL;
    CUSPARSE_CHECK(hipsparseSDDMM_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        &alpha, AMatDecsr, BMatDecsr, &beta, csrDescr, HIP_R_32F,
        HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize));
    CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));

    // run SDDMM
    CUSPARSE_CHECK(hipsparseSDDMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_TRANSPOSE, &alpha, AMatDecsr,
                                 BMatDecsr, &beta, csrDescr, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer));
    return values;
}

// vendor-provided cusparse by NVIDIA
int cusparse_sddmm_profile(
    torch::Tensor A, // A_ M x k
    torch::Tensor B, // B_ transpose N x k
    torch::Tensor rowPtr,
    torch::Tensor colIdx,
    torch::Tensor &values)
{
    const int M = A.size(0); // row of A
    const int K = A.size(1); // col of A
    const int N = B.size(0); // col of B
    const int nnz = values.size(0); 

    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t csrDescr;
    hipsparseDnMatDescr_t AMatDecsr, BMatDecsr;
    float alpha = 1.0f, beta = 0.0f;
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    // S
    auto csr_indptr_d = rowPtr.data_ptr<int>();
    auto csr_indices_d = colIdx.data_ptr<int>();
    auto csr_values_d = values.data_ptr<float>();
    // A & B
    auto A_d = A.data_ptr<float>();
    auto B_d = B.data_ptr<float>();

    // // creating sparse csr matrix
    CUSPARSE_CHECK(hipsparseCreateCsr(
        &csrDescr, M, N, nnz, csr_indptr_d, csr_indices_d, csr_values_d,
        HIPSPARSE_INDEX_32I, // index 32-integer for indptr
        HIPSPARSE_INDEX_32I, // index 32-integer for indices
        HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_32F // datatype: 32-bit float real number
        ));

    // creating dense matrices
    CUSPARSE_CHECK(hipsparseCreateDnMat(&AMatDecsr, M, K, K, A_d, HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&BMatDecsr, N, K, K, B_d, HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));

    // allocate workspace buffer
    size_t bufferSize = 0;
    void *dBuffer = NULL;
    CUSPARSE_CHECK(hipsparseSDDMM_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        &alpha, AMatDecsr, BMatDecsr, &beta, csrDescr, HIP_R_32F,
        HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize));
    CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < PROFILE; i++)
    {
        warmup<<<1024, 256>>>();
    }
    hipEventRecord(start, 0);
    for (int i = 0; i < PROFILE; i++)
    {
        // run SDDMM
        CUSPARSE_CHECK(hipsparseSDDMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     HIPSPARSE_OPERATION_TRANSPOSE, &alpha, AMatDecsr,
                                     BMatDecsr, &beta, csrDescr, HIP_R_32F,
                                     HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer));
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gflop = (float)nnz / 1e6 * K * 2;
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    printf("nnz: %d, K: %d, time: %f ms\n", nnz, K, milliseconds / PROFILE);
    return (int)(gflop / (milliseconds / PROFILE));
}